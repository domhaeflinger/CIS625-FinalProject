#include "hip/hip_runtime.h"
// Joshua Donnoe, Kyle Evens, and Dominik Haeflinger

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include "common.h"

#define NUM_THREADS 256

// from https://docs.nvidia.com/cuda/samples/6_Advanced/reduction/doc/reduction.pdf
// TODO edit to handle struct/tree
template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata, unsigned int n){
  extern __shared__ int sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockSize * 2) + tid;
  unsigned int gridSie = blockSize * 2 * gridDim.x;
  sdata[tid] = 0;

  while (i < n) { sdata[tid] += g_idata[i] + g_idata[i + blockSize]; i += gridSie; }
  __syncthreads();

  if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
  if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

  if(tid < 32) warpReduce(sdata, tid);
  if(tid == 0 ) g_odata[blockIdxx] = sdata[0];
}
// end from

// Calculates x position in matrix
__device__ void calcXPos(int adjIndex, int adjN, int *x){
  x = (int*) floor(adjN - sqrt(pow(adjN, 2) - adjIndex));
}

// Calculate the position in the matrix
__global__ void calcPosInMatrix(int index, int n, int *x, int *y){
  calcXPos(index * 2, n - (1/2), x);
}

// Calcuate edges between all points
__global__ void calculateEdge(){

}


// main duh
int main(int argc, char **argv) {

  hipDeviceSynchronize();

  if( find_option( argc, argv, "-h" ) >= 0 )
  {
      printf( "Options:\n" );
      printf( "-h to see this help\n" );
      printf( "-n <int> to set the number of particles\n" );
      printf( "-o <filename> to specify the output file name\n" );
      printf( "-s <filename> to specify the summary output file name\n" );
      return 0;
  }

  int n = read_int(argc, argv, "-n", 1000);

  char *savename = read_string(argc, argv, "-o", NULL);
  char *sumname = read_string(argc, argv, "-s", NULL);

  FILE *fsave = savename ? fopen(savename, "w") : NULL;
  FILE *fsum = sumname ? fopen(sumname, "a") : NULL;

  // GPU point data tructure
  edge_t * d_edges;
  hipMalloc((void **) &d_edges, n * (sizeof(point_t) + (n - 1) * sizeof(edge_t)));
  // GPU point data structure
  point_t * d_points = (point_t *)(((void *) d_edges) + (n * (n-1) * sizeof(edge_t)));

  double init_time = read_timer();
  // Initialize points
  hiprandGenerator_t gen; // Random number generator
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT); // Initialize generator
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL); // Set generator's seed
  hiprandGenerateUniform(gen, (float*)d_points, n); // Generate n random numbers in d_points

  // Initialize edges
  // TODO init edges

  hipDeviceSynchronize();
  init_time = read_timer() - init_time;
  double reduce_time = read_timer();

  // Calculate tree
  // TODO Calc tree

  hipDeviceSynchronize();
  reduce_time = read_timer() - reduce_time;

  printf("Initialization time = %g seconds\n", init_time);
  printf("n = %d, Reduction time = %g seconds\n", n, reduce_time);

  if (fsum)
  {
    fprintf(fsum, "%d %lf \n", n, reduce_time);
  }

  if (fsum)
  {
    fclose(fsum);
  }

  hipFree(d_edges);

  if (fsave)
  {
    fclose(fsave);
  }

  return 0;
}
