#include "hip/hip_runtime.h"
// Joshua Donnoe, Kyle Evens, and Dominik Haeflinger

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 256

// from https://docs.nvidia.com/cuda/samples/6_Advanced/reduction/doc/reduction.pdf
template <unsigned int blockSize>
__device__ void warpReduce(volatile int *sdata, unsigned int tid) {
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata, unsigned int n){
  extern __shared__ int sdata[];
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockSize * 2) + tid;
  unsigned int gridSie = blockSize * 2 * gridDim.x;
  sdata[tid] = 0;

  while (i < n) { sdata[tid] += g_idata[i] + g_idata[i + blockSize]; i += gridSie; }
  __syncthreads();

  if (blockSize >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
  if (blockSize >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
  if (blockSize >= 128) { if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads(); }

  if(tid < 32) warpReduce(sdata, tid);
  if(tid == 0 ) g_odata[blockIdxx] = sdata[0];
}
// end from

// generate graph/distance matrix


int main() {


  return 0;
}

/*
  struct edge
    int tree1
    int tree2
    double distance

*/
